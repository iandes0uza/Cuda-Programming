//-----------------------------------------------
//			ELEC374: Machine Problem 3.2
//					Ian DeSouza
//					 20232372
//				 20iagd@queensu.ca
//-----------------------------------------------

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <ctime>

void gpuMultiplication(float* inputA, float* inputB, float* result, int num);
void cpuMultiplication(float* inputA, float* inputB, float* result, int num);

int main() 
{
	const int arraySize = 5;
	int testSize [arraySize] = {125, 250, 500, 1000, 2000};
	for (int i = 0; i < arraySize; i++)
	{
		printf("*****************************************************************************\n");
		printf("------------------------------- %d X %d Matrix -------------------------------\n", testSize[i], testSize[i]);
		printf("*****************************************************************************\n");

		//Create Event Objects
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		float time = 0.0;

		//Allocate Pointer Memory
		size_t size = testSize[i] * testSize[i] * sizeof(float);
		float* matrixA = (float*)malloc(size);
		float* matrixB = (float*)malloc(size);
		float* cpuResult = (float*)malloc(size);
		float* gpuResult = (float*)malloc(size);

		//Allocate Device Memory
		int* devA;
		int* devB;
		int* devResult;
		hipMalloc((void**)&devA, size);
		hipMalloc((void**)&devB, size);
		hipMalloc((void**)&devResult, size);

		//Fill Matrix
		for (int x = 0; x < testSize[i]; x++)
		{
			for (int y = 0; y < testSize[i]; y++)
			{
				*(matrixA + x * testSize[i] + y) = (rand() % 100) / 10.0;
				*(matrixB + x * testSize[i] + y) = (rand() % 100) / 10.0;
				*(cpuResult + x * testSize[i] + y) = 0.0;
				*(gpuResult + x * testSize[i] + y) = 0.0;
			}
		}
		
		//Print Time (Host -> Device)
		hipEventRecord(start);
		hipMemcpy(devA, matrixA, size, hipMemcpyHostToDevice);
		hipMemcpy(devB, matrixB, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		printf("Elapsed Host -> Device:\t\t\t%0.2f \n", time);
		printf("------------------------------------------------------------------------------\n");

		//Print GPU Completion Time
		hipEventRecord(start);
		//gpuMultiplication <<< 1, 1 >>> (devA, devB, devResult, testSize[i]);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		printf("Elapsed GPU Computation:\t\t\t%0.2f \n", time);
		printf("------------------------------------------------------------------------------\n");

		//Print Time (Device -> Host)
		hipEventRecord(start);
		hipMemcpy(gpuResult, devResult, size, hipMemcpyDeviceToHost);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		printf("Elapsed Device -> Host:\t\t\t%0.2f \n", time);
		printf("------------------------------------------------------------------------------\n");
		
		//Print Time (Host -> Device)
		hipEventRecord(start);
		//cpuMultiplication();
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		printf("Elapsed CPU Computation -> Device:\t\t\t%0.2f \n", time);

		//Free Memory
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipFree(devA);
		hipFree(devB);
		free(matrixA);
		free(matrixB);
	}
}

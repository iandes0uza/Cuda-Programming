//-----------------------------------------------
//			ELEC374: Machine Problem 1
//					Ian DeSouza
//					 20232372
//				 20iagd@queensu.ca
//-----------------------------------------------

#include "hip/hip_runtime.h"

//This library will help us fetch necessary information



int main(void)
{
	//Declare the number of devices on the GPU Servers
	int numOfDev;
	hipGetDeviceCount(&numOfDev);

	//Iterate through each device & their properties
	for (int devNum = 0; devNum < numOfDev; devNum++)
	{
		//Call per device
	}
}

